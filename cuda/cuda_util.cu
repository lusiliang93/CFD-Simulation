#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "cuda_util.h"

#define THREADSPB 256

void cuda_init(int imax, int jmax){
}

#define get_index(i,j,jmax) ((jmax+2)*i+j)

__global__ void setbound_kernel_x(double* cudaDevice_u, double* cudaDevice_v, double* cudaDevice_u2, double* cudaDevice_v2, int imax, int jmax){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx>=1&&idx<jmax+1){
        cudaDevice_u2[get_index(idx, 0, jmax)] = 0;
        cudaDevice_u2[get_index(idx, imax, jmax)] = 0;
        cudaDevice_v2[get_index(idx, 0, jmax)] = -cudaDevice_v[get_index(idx, 1, jmax)];
        cudaDevice_v2[get_index(idx, imax+1, jmax)] = -cudaDevice_v[get_index(idx, imax, jmax)];
    }
}

__global__ void setbound_kernel_y(double* cudaDevice_u, double* cudaDevice_v, double* cudaDevice_u2, double* cudaDevice_v2, int imax, int jmax){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int us = 1;
    if(idx>=1&&idx<imax+1){
        cudaDevice_v2[get_index(0, idx, jmax)] = 0;
        cudaDevice_v2[get_index(jmax, idx, jmax)] = 0;
        cudaDevice_u2[get_index(0, idx, jmax)] = -cudaDevice_u[get_index(1, idx, jmax)];
        cudaDevice_u2[get_index(jmax+1, idx, jmax)] = 2*us - cudaDevice_u[get_index(jmax, idx, jmax)];
    }
}

void setbound(double **u,double **v,int imax,int jmax,int wW, int wE,int wN,int wS){
    double *cudaDevice_u, *cudaDevice_v, *cudaDevice_p, *cudaDevice_f, *cudaDevice_g, *cudaDevice_rhs;
    double *cudaDevice_u2, *cudaDevice_v2, *cudaDevice_p2, *cudaDevice_f2, *cudaDevice_g2, *cudaDevice_rhs2;
    hipMalloc(&cudaDevice_u, (imax+2)*(jmax+2) *sizeof(double));
    hipMalloc(&cudaDevice_v, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_p, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_f, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_g, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_rhs, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_u2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_v2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_p2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_f2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_g2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_rhs2, (imax+2)*(jmax+2)*sizeof(double));

    hipMemcpy(cudaDevice_u, u, sizeof(double)*(imax+2)*(jmax+2), hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_v, v, sizeof(double)*(imax+2)*(jmax+2), hipMemcpyHostToDevice);
    printf("ijmax: %d %d\n", imax, jmax);
    int nBlocks = (jmax+1 + THREADSPB-1)/THREADSPB;
    setbound_kernel_x<<<nBlocks, THREADSPB>>>(cudaDevice_u, cudaDevice_v, cudaDevice_u2, cudaDevice_v2,imax,jmax);
    nBlocks = (imax+1 + THREADSPB-1)/THREADSPB;
    setbound_kernel_y<<<nBlocks, THREADSPB>>>(cudaDevice_u, cudaDevice_v, cudaDevice_u2, cudaDevice_v2,imax,jmax);
    hipMemcpy(u, cudaDevice_u2, sizeof(double)*(imax+2)*(jmax+2), hipMemcpyDeviceToHost);
    hipMemcpy(v, cudaDevice_v2, sizeof(double)*(imax+2)*(jmax+2), hipMemcpyDeviceToHost);
    return;
}