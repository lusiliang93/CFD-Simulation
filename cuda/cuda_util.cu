#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "cuda_util.h"
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/extrema.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include "hipblas.h"
#include <float.h>

#define THREADSPB 256
#define get_index(m,n) ((jmax+2)*(m)+(n))

double *cudaDevice_u, *cudaDevice_v, *cudaDevice_p, *cudaDevice_f, *cudaDevice_g, *cudaDevice_rhs;
double *cudaDevice_u2, *cudaDevice_v2, *cudaDevice_p2, *cudaDevice_f2, *cudaDevice_g2, *cudaDevice_rhs2;

__global__ void fill_val(double* p, int length, int val){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    p[idx] = val;
}

__global__ void print_kernel(double* device_p,int imax, int jmax){
    int i,j;
    for(j=0;j<jmax+2;j++){
        for(i=0;i<imax+2;i++){
            // printf("%lf ", device_p[get_index(j,i)]);
        }
        // printf("\n");
    }
}

__global__ void sum_kernel(double* device_p, int length, double* device_sum){
    *device_sum = 0;
    for(int i=0;i<length;i++){
        *device_sum += device_p[i];
    }
}

/* sum all the value in vector device_p */
double sum_vector(double* device_p, int length){
    double* device_sum;
    hipMalloc(&device_sum, sizeof(double));
    sum_kernel<<<1, 1>>>(device_p, length, device_sum);
    double* tmp = (double*)malloc(sizeof(double));
    hipMemcpy(tmp,device_sum,sizeof(double),hipMemcpyDeviceToHost);
    double sum = *tmp;
    return sum;
}

__global__ void max_vector(double* device_p, int length, double* double_max){
    *double_max = -DBL_MAX;
    for(int i=0;i<length;i++){
        if(device_p[i]>*double_max){
            *double_max = device_p[i];
        }
    }
}

/* return the max value in vector device_p */
double max_vector(double* device_p, int length){
    double* device_max;
    hipMalloc(&device_max, sizeof(double));
    sum_kernel<<<1, 1>>>(device_p, length, device_max);
    double* tmp = (double*)malloc(sizeof(double));
    hipMemcpy(tmp,device_max,sizeof(double),hipMemcpyDeviceToHost);
    double mymax = *tmp;
    return mymax;
}

void cuda_init(int imax, int jmax){
    hipMalloc(&cudaDevice_u, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_v, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_p, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_f, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_g, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_rhs, (imax+2)*(jmax+2)*sizeof(double));

    hipMalloc(&cudaDevice_u2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_v2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_p2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_f2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_g2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_rhs2, (imax+2)*(jmax+2)*sizeof(double));

    int nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_u, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_v, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_p, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_f, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_g, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_rhs, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_u2, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_v2, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_p2, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_f2, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_g2, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_rhs2, (imax+2)*(jmax+2), 0);
}

/* copy from matrix to matrix 2(matrix 2 is the stale data from last iteration and is  read-only) */
void copy_matrix(int imax, int jmax){
}

double comp_delt(double* u, double* v, int imax, int jmax,double delx,double dely,double Re,double tau){
    double first,second,third,min;
    double delta = 1/(delx*delx)+1/(dely*dely);
    first = Re/2/delta;
    min=first;
    int length = (imax+2)*(jmax+2);
    
    hipMemcpy(u,cudaDevice_u2,(imax+2)*(jmax+2)*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(v,cudaDevice_v2,(imax+2)*(jmax+2)*sizeof(double),hipMemcpyDeviceToHost);

    double* result1 = thrust::max_element(thrust::host, u, u+length);
    double* result2 = thrust::max_element(thrust::host, v, v+length);
    second = delx/abs(*result1);
    third = dely/abs(*result2);
    if(min>second){
        min=second;
        if(min>third)
            min=third;
    }
    else{
        if(min>third)
            min=third;
    }
    double ret = tau*min;
    return ret;
}

__global__ void setbound_kernel(double* cudaDevice_u, double* cudaDevice_v, double* cudaDevice_u2, double* cudaDevice_v2, int imax, int jmax){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int j = idx;
    int i = idx;
    int us = 1;
    if(j>=1&&j<jmax+1){
        cudaDevice_u[get_index(j, 0)] = 0;
        cudaDevice_u[get_index(j, imax)] = 0;
        cudaDevice_v[get_index(j, 0)] = -cudaDevice_v2[get_index(j, 1)];
        cudaDevice_v[get_index(j, imax+1)] = -cudaDevice_v2[get_index(j, imax)];
    }
    if(i>=1&&i<imax+1){
        cudaDevice_v[get_index(0, i)] = 0;
        cudaDevice_v[get_index(jmax, i)] = 0;
        cudaDevice_u[get_index(0, i)] = -cudaDevice_u2[get_index(1, i)];
        cudaDevice_u[get_index(jmax+1, i)] = 2*us-cudaDevice_u2[get_index(jmax, i)];
    }
}

void setbound(int imax,int jmax,int wW, int wE,int wN,int wS){
    int nBlocks = (max(jmax+1,imax+1) + THREADSPB-1)/THREADSPB;
    setbound_kernel<<<nBlocks, THREADSPB>>>(cudaDevice_u, cudaDevice_v, cudaDevice_u2, cudaDevice_v2, imax, jmax);
    double* tmp_u = cudaDevice_u2;
    cudaDevice_u2 = cudaDevice_u;
    cudaDevice_u = tmp_u;
    double* tmp_v = cudaDevice_v2;
    cudaDevice_v2 = cudaDevice_v;
    cudaDevice_v = tmp_v;
    hipDeviceSynchronize();
    return;
}

__global__ void init_uvp_kernel(double* cudaDevice_u, double* cudaDevice_v, double* cudaDevice_p, int imax, int jmax, int UI, int VI, int PI){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int j = idx/(jmax+2);
    int i = idx%(jmax+2);
    if(idx<(imax+2)*(jmax+2)){
        cudaDevice_u[get_index(j,i)] = UI;
        cudaDevice_v[get_index(j,i)] = VI;
        cudaDevice_p[get_index(j,i)] = PI;
    }
}

void init_uvp(int imax, int jmax,int UI, int VI, int PI){
    int nBlocks = ((jmax+2)*(imax+2) + THREADSPB-1)/THREADSPB;
    init_uvp_kernel<<<nBlocks, THREADSPB>>>(cudaDevice_u, cudaDevice_v, cudaDevice_p, imax,jmax,UI,VI,PI);
    hipDeviceSynchronize();
}

__global__ void comp_fg_kernel_1(double* cudaDevice_u2, double* cudaDevice_v2, double* cudaDevice_f, double* cudaDevice_g, int imax, int jmax){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = idx;
    int j = idx;
    if(j>=1&&j<jmax+1){
        cudaDevice_f[get_index(j,0)] = cudaDevice_u2[get_index(j,0)];
        cudaDevice_f[get_index(j,imax)] = cudaDevice_u2[get_index(j,imax)];
    }
    if(i>=1&&i<imax+1){
        cudaDevice_g[get_index(0,i)] = cudaDevice_v2[get_index(0,i)];
        cudaDevice_g[get_index(jmax,i)] = cudaDevice_v2[get_index(jmax,i)];
    }
}

__global__ void comp_fg_kernel_2(double* cudaDevice_u2, double* cudaDevice_v2, double* cudaDevice_f, double* cudaDevice_g, int imax, int jmax, double delt,double delx,double dely,double gx,double gy,double gamma,double Re){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int j = idx/(jmax+2);
    int i = idx%(jmax+2);
    double a,b,c,d,e,ff,gg,h,va,vb,u2x,uvy,u2x2,u2y2;
    double ua,ub,uvx,v2y,v2x2,v2y2;
    if(j>=1&&j<jmax+1){
        if(i>=1&&i<imax){
            a = cudaDevice_u2[get_index(j,i)] + cudaDevice_u2[get_index(j,i+1)];
            b = cudaDevice_u2[get_index(j,i-1)] + cudaDevice_u2[get_index(j,i)];
            c = cudaDevice_u2[get_index(j,i)] - cudaDevice_u2[get_index(j,i+1)];
            d = cudaDevice_u2[get_index(j,i-1)] - cudaDevice_u2[get_index(j,i)];
            e = cudaDevice_u2[get_index(j,i)] + cudaDevice_u2[get_index(j+1,i)];
            ff = cudaDevice_u2[get_index(j-1,i)] + cudaDevice_u2[get_index(j,i)];
            gg = cudaDevice_u2[get_index(j,i)] - cudaDevice_u2[get_index(j+1,i)];
            h = cudaDevice_u2[get_index(j-1,i)] - cudaDevice_u2[get_index(j,i)];
            va = cudaDevice_v2[get_index(j,i)] + cudaDevice_v2[get_index(j,i+1)];
            vb = cudaDevice_v2[get_index(j-1,i)] + cudaDevice_v2[get_index(j-1,i+1)];
            u2x = 1/delx*((a/2)*(a/2)-(b/2)*(b/2))+gamma*1/delx*(abs(a)/2*c/2-abs(b)/2*d/2);
            uvy = 1/dely*(va/2*e/2-vb/2*ff/2)+gamma*1/dely*(abs(va)/2*gg/2-abs(vb)/2*h/2);
            u2x2 = (cudaDevice_u2[get_index(j,i+1)] - 2*cudaDevice_u2[get_index(j,i)] + cudaDevice_u2[get_index(j,i-1)])/(delx*delx);
            u2y2 = (cudaDevice_u2[get_index(j+1,i)] - 2*cudaDevice_u2[get_index(j,i)] + cudaDevice_u2[get_index(j-1,i)])/(dely*dely);
            cudaDevice_f[get_index(j,i)] = cudaDevice_u2[get_index(j,i)] + delt*(1/Re*(u2x2+u2y2)-u2x-uvy+gx);
        }
    }

    if(j>=1&&j<jmax){
        if(i>=1&&i<imax+1){
            a = cudaDevice_v2[get_index(j,i)] + cudaDevice_v2[get_index(j,i+1)];
            b = cudaDevice_v2[get_index(j,i-1)] + cudaDevice_v2[get_index(j,i)];
            c = cudaDevice_v2[get_index(j,i)] - cudaDevice_v2[get_index(j,i+1)];
            d = cudaDevice_v2[get_index(j,i-1)] - cudaDevice_v2[get_index(j,i)];
            e = cudaDevice_v2[get_index(j,i)] + cudaDevice_v2[get_index(j+1,i)];
            ff = cudaDevice_v2[get_index(j-1,i)] + cudaDevice_v2[get_index(j,i)];
            gg = cudaDevice_v2[get_index(j,i)] - cudaDevice_v2[get_index(j+1,i)];
            h = cudaDevice_v2[get_index(j-1,i)] - cudaDevice_v2[get_index(j,i)];
            ua = cudaDevice_u2[get_index(j,i)] + cudaDevice_u2[get_index(j+1,i)];
            ub = cudaDevice_u2[get_index(j,i-1)] + cudaDevice_u2[get_index(j+1,i-1)];
            uvx = 1/delx*(ua/2*a/2-ub/2*b/2)+gamma*1/delx*(abs(ua)/2*c/2-abs(ub)/2*d/2);
            v2y = 1/dely*((e/2)*(e/2)-(ff/2)*(ff/2))+gamma*1/dely*(abs(e)/2*gg/2-abs(ff)/2*h/2);
            v2x2 = (cudaDevice_v2[get_index(j,i+1)] - 2*cudaDevice_v2[get_index(j,i)] + cudaDevice_v2[get_index(j,i-1)])/(delx*delx);
            v2y2 = (cudaDevice_v2[get_index(j+1,i)] - 2*cudaDevice_v2[get_index(j,i)] + cudaDevice_v2[get_index(j-1,i)])/(dely*dely);
            cudaDevice_g[get_index(j,i)] = cudaDevice_v2[get_index(j,i)] + delt*(1/Re*(v2x2+v2y2)-uvx-v2y+gy);
        }
    }
}

void comp_fg(int imax, int jmax,double delt,double delx,double dely,double gx,double gy,double gamma,double Re){
    int nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_f, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_g, (imax+2)*(jmax+2), 0);

    nBlocks = (max(imax,jmax)+2 + THREADSPB-1)/THREADSPB;
    comp_fg_kernel_1<<<nBlocks, THREADSPB>>>(cudaDevice_u2, cudaDevice_v2, cudaDevice_f, cudaDevice_g, imax, jmax);

    nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
    comp_fg_kernel_2<<<nBlocks, THREADSPB>>>(cudaDevice_u2, cudaDevice_v2, cudaDevice_f, cudaDevice_g, imax, jmax, delt, delx, dely, gx, gy, gamma, Re);

    double* tmp_f = cudaDevice_f2;
    cudaDevice_f2 = cudaDevice_f;
    cudaDevice_f = tmp_f;
    double* tmp_g = cudaDevice_g2;
    cudaDevice_g2 = cudaDevice_g;
    cudaDevice_g = tmp_g;

    hipDeviceSynchronize();
}

__global__ void comp_rhs_kernel(double* cudaDevice_f2, double* cudaDevice_g2, double* cudaDevice_rhs, int imax, int jmax, double delx, double dely, double delt){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int j = idx/(jmax+2);
    int i = idx%(jmax+2);
    if(j>=1&&j<jmax+1){
        if(i>=1&&i<imax+1){
            double tmp = (cudaDevice_f2[get_index(j,i)]-cudaDevice_f2[get_index(j,i-1)])/delx + (cudaDevice_g2[get_index(j,i)]-cudaDevice_g2[get_index(j-1,i)])/dely;
            cudaDevice_rhs[get_index(j,i)] = 1/delt * tmp;
        }
    }
}

void comp_rhs(int imax, int jmax,double delt,double delx,double dely){
    int nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_rhs, (imax+2)*(jmax+2), 0);

    nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
    comp_rhs_kernel<<<nBlocks, THREADSPB>>>(cudaDevice_f2, cudaDevice_g2, cudaDevice_rhs, imax, jmax, delx, dely, delt);

    double* tmp_rhs = cudaDevice_rhs2;
    cudaDevice_rhs2 = cudaDevice_rhs;
    cudaDevice_rhs = tmp_rhs;

    hipDeviceSynchronize();
}

__global__ void poisson_kernel_1(double* cudaDevice_p, double* cudaDevice_p2, double* cudaDevice_r, int imax, int jmax){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int j = idx;
    int i = idx;
    if(j>=1&&j<jmax+1){
        cudaDevice_p[get_index(j,0)] = cudaDevice_p[get_index(j,1)];
        cudaDevice_p[get_index(j,imax+1)] = cudaDevice_p[get_index(j,imax)];
    }
    if(i>=1&&i<imax+1){
        cudaDevice_p[get_index(0,i)] = cudaDevice_p[get_index(1,i)];
        cudaDevice_p[get_index(jmax+1,i)] = cudaDevice_p[get_index(jmax,i)];
    }
}

__global__ void poisson_kernel_serial(double* cudaDevice_r, double* cudaDevice_p, double* cudaDevice_p2, double* cudaDevice_rhs2, int imax, int jmax, double delx, double dely, double omg, int mymod){
    int i,j;
    int eiw,eie,ejn,ejs;
    for(j=1;j<jmax+1;j++){
        for(i=1;i<imax+1;i++){
            eiw=1;eie=1;ejs=1;ejn=1;
            double a1 = (1-omg)*cudaDevice_p[get_index(j,i)];
            double a2 = omg/((eie+eiw)/(delx*delx)+(ejn+ejs)/(dely*dely));
            double aa1 = eie*cudaDevice_p[get_index(j,i+1)];
            double aa2 = eiw*cudaDevice_p[get_index(j,i-1)];
            double a4 = (aa1+aa2)/(delx*delx);
            double aa3 = ejn*cudaDevice_p[get_index(j+1,i)];
            double aa4 = ejs*cudaDevice_p[get_index(j-1,i)];
            double a5 = (aa3+aa4)/(dely*dely);
            double a6 = cudaDevice_rhs2[get_index(j,i)];
            double a3 = (a4+a5-a6);
            cudaDevice_p[get_index(j,i)] = a1 + a2 * a3;

            cudaDevice_r[get_index(j,i)] = (
                eie*(cudaDevice_p[get_index(j,i+1)]-cudaDevice_p[get_index(j,i)])
                -eiw*(cudaDevice_p[get_index(j,i)]-cudaDevice_p[get_index(j,i-1)])
                )/(delx*delx)
            +    (
                ejn*(cudaDevice_p[get_index(j+1,i)]-cudaDevice_p[get_index(j,i)])
                -ejs*(cudaDevice_p[get_index(j,i)]-cudaDevice_p[get_index(j-1,i)])
                )/(dely*dely)
            - cudaDevice_rhs2[get_index(j,i)];

            cudaDevice_r[get_index(j,i)] = cudaDevice_r[get_index(j,i)]*cudaDevice_r[get_index(j,i)];
        }
    }
}

__global__ void poisson_kernel_odd_even(double* cudaDevice_r, double* cudaDevice_p, double* cudaDevice_p2, double* cudaDevice_rhs2, int imax, int jmax, double delx, double dely, double omg, int mymod){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int j = idx/(jmax+2);
    int i = idx%(jmax+2);
    int eiw,eie,ejn,ejs;
    if(j>=1&&j<jmax+1){
        if(i>=1&&i<imax+1){
            if((i+j)%2==mymod){
                eiw=1;eie=1;ejs=1;ejn=1;
                double a1 = (1-omg)*cudaDevice_p[get_index(j,i)];
                double a2 = omg/((eie+eiw)/(delx*delx)+(ejn+ejs)/(dely*dely));
                double aa1 = eie*cudaDevice_p[get_index(j,i+1)];
                double aa2 = eiw*cudaDevice_p[get_index(j,i-1)];
                double a4 = (aa1+aa2)/(delx*delx);
                double aa3 = ejn*cudaDevice_p[get_index(j+1,i)];
                double aa4 = ejs*cudaDevice_p[get_index(j-1,i)];
                double a5 = (aa3+aa4)/(dely*dely);
                double a6 = cudaDevice_rhs2[get_index(j,i)];
                double a3 = (a4+a5-a6);
                cudaDevice_p[get_index(j,i)] = a1 + a2 * a3;

                cudaDevice_r[get_index(j,i)] = (
                    eie*(cudaDevice_p[get_index(j,i+1)]-cudaDevice_p[get_index(j,i)])
                    -eiw*(cudaDevice_p[get_index(j,i)]-cudaDevice_p[get_index(j,i-1)])
                    )/(delx*delx)
                +    (
                    ejn*(cudaDevice_p[get_index(j+1,i)]-cudaDevice_p[get_index(j,i)])
                    -ejs*(cudaDevice_p[get_index(j,i)]-cudaDevice_p[get_index(j-1,i)])
                    )/(dely*dely)
                - cudaDevice_rhs2[get_index(j,i)];

                cudaDevice_r[get_index(j,i)] = cudaDevice_r[get_index(j,i)]*cudaDevice_r[get_index(j,i)];
            }
            else{
                cudaDevice_p[get_index(j,i)] = cudaDevice_p[get_index(j,i)];
            }
        }
    }
}

int poisson_serial(int imax, int jmax,double delx,double dely,double eps,int itermax,double omg){
    int it;
    double sum;
    double res;
    double* cudaDevice_r;
    hipMalloc(&cudaDevice_r, (imax+2)*(jmax+2) *sizeof(double));
    int nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_r, (imax+2)*(jmax+2), 0);
    for(it=0;it<itermax;it++){
        nBlocks = (max(imax,jmax)+2 + THREADSPB-1)/THREADSPB;
        poisson_kernel_1<<<nBlocks, THREADSPB>>>(cudaDevice_p, cudaDevice_p2, cudaDevice_r, imax, jmax);
        hipDeviceSynchronize();

        // serial part
        nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
        poisson_kernel_serial<<<1,1>>>(cudaDevice_r, cudaDevice_p, cudaDevice_p2, cudaDevice_rhs2, imax, jmax, delx, dely, omg, 1);
        hipDeviceSynchronize();

        // double* sum_arr = (double*)malloc(sizeof(double)*(imax+2)*(jmax+2));
        //hipMemcpy(sum_arr,cudaDevice_r,(imax+2)*(jmax+2)*sizeof(double),hipMemcpyDeviceToHost);
        // sum = thrust::reduce(thrust::host, sum_arr, sum_arr + (imax+2)*(jmax+2));
        
        // sum = sum_vector(cudaDevice_r, (imax+2)*(jmax+2));
        // res=sqrt(sum/(imax*jmax));
        // if(res<eps){
           // break;
        // }
        /* copy p to stale p(p2) */
        // double* tmp_p = cudaDevice_p2;
        // cudaDevice_p2 = cudaDevice_p;
        // cudaDevice_p = tmp_p;
        hipDeviceSynchronize();
    }
    hipFree(cudaDevice_r);
    return it;
}

int poisson(int imax, int jmax,double delx,double dely,double eps,int itermax,double omg){
    int it;
    double sum;
    double res;
    double* cudaDevice_r;
    hipMalloc(&cudaDevice_r, (imax+2)*(jmax+2) *sizeof(double));
    int nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_r, (imax+2)*(jmax+2), 0);
    for(it=0;it<itermax;it++){
        nBlocks = (max(imax,jmax)+2 + THREADSPB-1)/THREADSPB;
        poisson_kernel_1<<<nBlocks, THREADSPB>>>(cudaDevice_p, cudaDevice_p2, cudaDevice_r, imax, jmax);
        hipDeviceSynchronize();

        // red black parallelization
        nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
        poisson_kernel_odd_even<<<nBlocks, THREADSPB>>>(cudaDevice_r, cudaDevice_p, cudaDevice_p2, cudaDevice_rhs2, imax, jmax, delx, dely, omg, 1);
        hipDeviceSynchronize();

        poisson_kernel_odd_even<<<nBlocks, THREADSPB>>>(cudaDevice_r, cudaDevice_p, cudaDevice_p2, cudaDevice_rhs2, imax, jmax, delx, dely, omg, 0);
        hipDeviceSynchronize();

        sum = sum_vector(cudaDevice_r, (imax+2)*(jmax+2));
        res=sqrt(sum/(imax*jmax));
        if(res<eps){
            break;
        }
        /* copy p to stale p(p2) */
        hipDeviceSynchronize();
    }
    hipFree(cudaDevice_r);
    return it;
}
__global__ void adap_uv_kernel(double* cudaDevice_u, double* cudaDevice_v, double* cudaDevice_f2, double* cudaDevice_g2, double* cudaDevice_p2, int imax,int jmax,double delt,double delx,double dely){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int j = idx/(jmax+2);
    int i = idx%(jmax+2);
    if(j>=1&&j<jmax+1){
        if(i>=1&&i<imax){
            cudaDevice_u[get_index(j,i)] = cudaDevice_f2[get_index(j,i)]-delt/delx*(cudaDevice_p2[get_index(j,i+1)]-cudaDevice_p2[get_index(j,i)]);
        }
    }
    if(j>=1&&j<jmax){
        if(i>=1&&i<imax+1){
            cudaDevice_v[get_index(j,i)] = cudaDevice_g2[get_index(j,i)]-delt/dely*(cudaDevice_p2[get_index(j+1,i)]-cudaDevice_p2[get_index(j,i)]);
        }
    }
}

void adap_uv(int imax, int jmax, double delt, double delx, double dely){
    int nBlocks = nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
    adap_uv_kernel<<<nBlocks, THREADSPB>>>(cudaDevice_u, cudaDevice_v, cudaDevice_f2, cudaDevice_g2, cudaDevice_p, imax, jmax, delt, delx, dely);

    double* tmp_u = cudaDevice_u2;
    cudaDevice_u2 = cudaDevice_u;
    cudaDevice_u = tmp_u;
    double* tmp_v = cudaDevice_v2;
    cudaDevice_v2 = cudaDevice_v;
    cudaDevice_v = tmp_v;
}

void get_data(double* u, double* v, int imax, int jmax){
    hipMemcpy(u,cudaDevice_u2,(imax+2)*(jmax+2)*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(v,cudaDevice_v2,(imax+2)*(jmax+2)*sizeof(double),hipMemcpyDeviceToHost);
}