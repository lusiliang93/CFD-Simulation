#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <math.h>
#include "init.h"
#include "uvp.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

__global__ void setbound_kernel_x(double** u, double** v, int imax, int jmax){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx>=1&&idx<jmax+1){
        u[idx][0] = 0;
        u[idx][imax] = 0;
        v[idx][0] = -v[idx][1];
        v[idx][imax+1] = -v[idx][imax];
    }
}

__global__ void setbound_kernel_y(double** u, double** v, int imax, int jmax){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int us = 1;
    if(idx>=1&&idx<imax+1){
        v[0][idx] = 0;
        v[jmax][idx] = 0;
        u[0][idx] = -u[1][idx];
        u[jmax+1][idx] = 2*us - u[jmax][idx];
    }
}

void setbound(double **u,double **v,int imax,int jmax,int wW, int wE,int wN,int wS){
    int nBlocks  = (jmax+1 + THREADSPB-1)/THREADSPB;
    setbound_kernel_x<<<nBlocks, THREADSPB>>>(u,v,imax,jmax);
    nBlocks  = (imax+1 + THREADSPB-1)/THREADSPB;
    setbound_kernel_y<<<nBlocks, THREADSPB>>>(u,v,imax,jmax);
    return;
}