#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "cuda_util.h"
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/extrema.h>
#include <thrust/reduce.h>
#include "hipblas.h"

#define THREADSPB 256
#define get_index(i,j) ((jmax+2)*i+j)

double *cudaDevice_u, *cudaDevice_v, *cudaDevice_p, *cudaDevice_f, *cudaDevice_g, *cudaDevice_rhs;
double *cudaDevice_u2, *cudaDevice_v2, *cudaDevice_p2, *cudaDevice_f2, *cudaDevice_g2, *cudaDevice_rhs2;

__global__ void fill_val(double* p, int length, int val){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    p[idx] = val;
}

/* sum all the value in vector device_p */
double sum_vector(double* device_p, int length){
    thrust::device_ptr<double> d_ptr = thrust::device_pointer_cast(device_p);
    double sum = thrust::reduce(d_ptr, d_ptr+length, (double)0.0, thrust::plus<double>());
    return sum;
}

/* return the max value in vector device_p */
double max_vector(double* device_p, int length){
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    double mymax = 0.0;
    int max_idx = 0;
    hipblasIdamax(handle, length, device_p, 1, &max_idx);
    mymax = device_p[max_idx];
    return mymax;
}

void cuda_init(int imax, int jmax){
    hipMalloc(&cudaDevice_u, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_v, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_p, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_f, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_g, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_rhs, (imax+2)*(jmax+2)*sizeof(double));

    hipMalloc(&cudaDevice_u2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_v2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_p2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_f2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_g2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_rhs2, (imax+2)*(jmax+2)*sizeof(double));

    int nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_u, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_v, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_p, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_f, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_g, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_rhs, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_u2, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_v2, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_p2, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_f2, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_g2, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_rhs2, (imax+2)*(jmax+2), 0);
}

/* copy from matrix to matrix 2(matrix 2 is the stale data from last iteration and is  read-only) */
void copy_matrix(int imax, int jmax){
    double* tmp_u = cudaDevice_u2;
    double* tmp_v = cudaDevice_v2;
    double* tmp_p = cudaDevice_p2;
    double* tmp_f = cudaDevice_f2;
    double* tmp_g = cudaDevice_g2;
    double* tmp_rhs = cudaDevice_rhs2;
    cudaDevice_u2 = cudaDevice_u;
    cudaDevice_v2 = cudaDevice_v;
    cudaDevice_p2 = cudaDevice_p;
    cudaDevice_f2 = cudaDevice_f;
    cudaDevice_g2 = cudaDevice_g;
    cudaDevice_rhs2 = cudaDevice_rhs;
    cudaDevice_u = tmp_u;
    cudaDevice_v = tmp_v;
    cudaDevice_p = tmp_p;
    cudaDevice_f = tmp_f;
    cudaDevice_g = tmp_g;
    cudaDevice_rhs = tmp_rhs;
    int nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_u, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_v, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_p, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_f, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_g, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_rhs, (imax+2)*(jmax+2), 0);
}

double comp_delt(int imax, int jmax,double delx,double dely,double Re,double tau){
    double first,second,third,min;
    double delta = 1/(delx*delx)+1/(dely*dely);
    first = Re/2/delta;
    min=first;
    int length = (imax+2)*(jmax+2);
    second = delx/abs(max_vector(cudaDevice_u2, length));
    third= dely/abs(max_vector(cudaDevice_v2, length));
    if(min>second){
        min=second;
        if(min>third)
            min=third;
    }
    else{
        if(min>third)
            min=third;
    }
    double ret = tau*min;
    return ret;
}

__global__ void setbound_kernel_x(double* cudaDevice_u, double* cudaDevice_v, double* cudaDevice_u2, double* cudaDevice_v2, int imax, int jmax){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int j = idx;
    if(idx>=1&&idx<jmax+1){
        cudaDevice_u[get_index(j, 0)] = 0;
        cudaDevice_u[get_index(j, imax)] = 0;
        cudaDevice_v[get_index(j, 0)] = -cudaDevice_v2[get_index(j, 1)];
        cudaDevice_v[get_index(j, imax+1)] = -cudaDevice_v2[get_index(j, imax)];
    }
}

__global__ void setbound_kernel_y(double* cudaDevice_u, double* cudaDevice_v, double* cudaDevice_u2, double* cudaDevice_v2, int imax, int jmax){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = idx;
    int us = 1;
    if(idx>=1&&idx<imax+1){
        cudaDevice_v[get_index(0, i)] = 0;
        cudaDevice_v[get_index(jmax, i)] = 0;
        cudaDevice_u[get_index(0, i)] = -cudaDevice_u2[get_index(1, i)];
        cudaDevice_u[get_index(jmax+1, i)] = 2*us - cudaDevice_u2[get_index(jmax, i)];
    }
}

void setbound(int imax,int jmax,int wW, int wE,int wN,int wS){
    int nBlocks = (jmax+1 + THREADSPB-1)/THREADSPB;
    setbound_kernel_x<<<nBlocks, THREADSPB>>>(cudaDevice_u, cudaDevice_v, cudaDevice_u2, cudaDevice_v2,imax,jmax);
    nBlocks = (imax+1 + THREADSPB-1)/THREADSPB;
    setbound_kernel_y<<<nBlocks, THREADSPB>>>(cudaDevice_u, cudaDevice_v, cudaDevice_u2, cudaDevice_v2,imax,jmax);
    return;
}

__global__ void init_uvp_kernel(double* cudaDevice_u, double* cudaDevice_v, double* cudaDevice_p, int imax, int jmax, int UI, int VI, int PI){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int j = idx/(jmax+2);
    int i = idx%(jmax+2);
    if(idx<(imax+2)*(jmax+2)){
        cudaDevice_u[get_index(j,i)] = UI;
        cudaDevice_v[get_index(j,i)] = VI;
        cudaDevice_p[get_index(j,i)] = PI;
    }
}

void init_uvp(int imax, int jmax,int UI, int VI, int PI){
    int nBlocks = ((jmax+2)*(imax+2) + THREADSPB-1)/THREADSPB;
    init_uvp_kernel<<<nBlocks, THREADSPB>>>(cudaDevice_u, cudaDevice_v, cudaDevice_p, imax,jmax,UI,VI,PI);
}

__global__ void comp_fg_kernel_1(double* cudaDevice_u2, double* cudaDevice_v2, double* cudaDevice_f, double* cudaDevice_g, int imax, int jmax){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = idx;
    int j = idx;
    if(j>=1&&j<jmax+1){
        cudaDevice_f[get_index(j,0)] = cudaDevice_u2[get_index(j,0)];
        cudaDevice_f[get_index(j,imax)] = cudaDevice_u2[get_index(j,imax)];
    }
    if(i>=1&&i<imax+1){
        cudaDevice_g[get_index(0,i)] = cudaDevice_v2[get_index(0,i)];
        cudaDevice_g[get_index(jmax,i)] = cudaDevice_v2[get_index(jmax,i)];
    }
}

__global__ void comp_fg_kernel_2(double* cudaDevice_u2, double* cudaDevice_v2, double* cudaDevice_f, double* cudaDevice_g, int imax, int jmax, double delt,double delx,double dely,double gx,double gy,double gamma,double Re){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int j = idx/(jmax+2);
    int i = idx%(jmax+2);
    double a,b,c,d,e,ff,gg,h,va,vb,u2x,uvy,u2x2,u2y2;
    double ua,ub,uvx,v2y,v2x2,v2y2;
    if(j>=1&&j<jmax+1){
        if(i>=1&&i<imax){
            a = cudaDevice_u2[get_index(j,i)] + cudaDevice_u2[get_index(j,i+1)];
            b = cudaDevice_u2[get_index(j,i-1)] + cudaDevice_u2[get_index(j,i)];
            c = cudaDevice_u2[get_index(j,i)] - cudaDevice_u2[get_index(j,i+1)];
            d = cudaDevice_u2[get_index(j,i-1)] - cudaDevice_u2[get_index(j,i)];
            e = cudaDevice_u2[get_index(j,i)] + cudaDevice_u2[get_index(j+1,i)];
            ff = cudaDevice_u2[get_index(j-1,i)] + cudaDevice_u2[get_index(j,i)];
            gg = cudaDevice_u2[get_index(j,i)] - cudaDevice_u2[get_index(j+1,i)];
            h = cudaDevice_u2[get_index(j-1,i)] - cudaDevice_u2[get_index(j,i)];
            va = cudaDevice_v2[get_index(j,i)] + cudaDevice_v2[get_index(j,i+1)];
            vb = cudaDevice_v2[get_index(j-1,i)] + cudaDevice_v2[get_index(j-1,i+1)];
            u2x = 1/delx*((a/2)*(a/2)-(b/2)*(b/2))+gamma*1/delx*(abs(a)/2*c/2-abs(b)/2*d/2);
            uvy = 1/dely*(va/2*e/2-vb/2*ff/2)+gamma*1/dely*(abs(va)/2*gg/2-abs(vb)/2*h/2);
            u2x2 = (cudaDevice_u2[get_index(j,i+1)] - 2*cudaDevice_u2[get_index(j,i)] + cudaDevice_u2[get_index(j,i-1)])/(delx*delx);
            u2y2 = (cudaDevice_u2[get_index(j+1,i)] - 2*cudaDevice_u2[get_index(j,i)] + cudaDevice_u2[get_index(j-1,i)])/(dely*dely);
            cudaDevice_f[get_index(j,i)] = cudaDevice_u2[get_index(j,i)] + delt*(1/Re*(u2x2+u2y2)-u2x-uvy+gx);
        }
    }

    if(j>=1&&j<jmax){
        if(i>=1&&i<imax+1){
            a = cudaDevice_v2[get_index(j,i)] + cudaDevice_v2[get_index(j,i+1)];
            b = cudaDevice_v2[get_index(j,i-1)] + cudaDevice_v2[get_index(j,i)];
            c = cudaDevice_v2[get_index(j,i)] - cudaDevice_v2[get_index(j,i+1)];
            d = cudaDevice_v2[get_index(j,i-1)] - cudaDevice_v2[get_index(j,i)];
            e = cudaDevice_v2[get_index(j,i)] + cudaDevice_v2[get_index(j+1,i)];
            ff = cudaDevice_v2[get_index(j-1,i)] + cudaDevice_v2[get_index(j,i)];
            gg = cudaDevice_v2[get_index(j,i)] - cudaDevice_v2[get_index(j+1,i)];
            h = cudaDevice_v2[get_index(j-1,i)] - cudaDevice_v2[get_index(j,i)];
            ua = cudaDevice_u2[get_index(j,i)] + cudaDevice_u2[get_index(j+1,i)];
            ub = cudaDevice_u2[get_index(j,i-1)] + cudaDevice_u2[get_index(j+1,i-1)];
            uvx = 1/delx*(ua/2*a/2-ub/2*b/2)+gamma*1/delx*(abs(ua)/2*c/2-abs(ub)/2*d/2);
            v2y = 1/dely*((e/2)*(e/2)-(ff/2)*(ff/2))+gamma*1/dely*(abs(e)/2*gg/2-abs(ff)/2*h/2);
            v2x2 = (cudaDevice_v2[get_index(j,i+1)] - 2*cudaDevice_v2[get_index(j,i)] + cudaDevice_v2[get_index(j,i-1)])/(delx*delx);
            v2y2 = (cudaDevice_v2[get_index(j+1,i)] - 2*cudaDevice_v2[get_index(j,i)] + cudaDevice_v2[get_index(j-1,i)])/(dely*dely);
            cudaDevice_g[get_index(j,i)] = cudaDevice_v2[get_index(j,i)] + delt*(1/Re*(v2x2+v2y2)-uvx-v2y+gy);
        }
    }
}

void comp_fg(int imax, int jmax,double delt,double delx,double dely,double gx,double gy,double gamma,double Re){
    int nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_f, (imax+2)*(jmax+2), 0);
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_g, (imax+2)*(jmax+2), 0);

    nBlocks = (max(imax,jmax)+2 + THREADSPB-1)/THREADSPB;
    comp_fg_kernel_1<<<nBlocks, THREADSPB>>>(cudaDevice_u2, cudaDevice_v2, cudaDevice_f, cudaDevice_g, imax, jmax);

    nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
    comp_fg_kernel_2<<<nBlocks, THREADSPB>>>(cudaDevice_u2, cudaDevice_v2, cudaDevice_f, cudaDevice_g, imax, jmax, delt, delx, dely, gx, gy, gamma, Re);
}

__global__ void comp_rhs_kernel(double* cudaDevice_f2, double* cudaDevice_g2, double* cudaDevice_rhs, int imax, int jmax, double delx, double dely, double delt){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int j = idx/(jmax+2);
    int i = idx%(jmax+2);
    if(j>=1&&j<jmax+1){
        if(i>=1&&i<imax+1){
            int tmp = (cudaDevice_f2[get_index(j,i)]-cudaDevice_f2[get_index(j,i-1)])/delx + (cudaDevice_g2[get_index(j,i)]-cudaDevice_g2[get_index(j-1,i)])/dely;
            cudaDevice_rhs[get_index(j,i)] = 1/delt * tmp;
        }
    }
}

void comp_rhs(int imax, int jmax,double delt,double delx,double dely){
    int nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
    fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_rhs, (imax+2)*(jmax+2), 0);

    nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
    comp_rhs_kernel<<<nBlocks, THREADSPB>>>(cudaDevice_f2, cudaDevice_g2, cudaDevice_rhs, imax, jmax, delx, dely, delt);
}

__global__ void poisson_kernel_1(double* cudaDevice_p, double* cudaDevice_p2, double* cudaDevice_r, int imax, int jmax){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int j = idx;
    int i = idx;
    if(j>=1&&j<jmax+1){
        cudaDevice_p[get_index(j,0)] = cudaDevice_p2[get_index(j,1)];
        cudaDevice_p[get_index(j,imax+1)] = cudaDevice_p2[get_index(j,imax)];
    }
    if(i>=1&&i<imax+1){
        cudaDevice_p[get_index(0,i)] = cudaDevice_p2[get_index(1,i)];
        cudaDevice_p[get_index(jmax+1,i)] = cudaDevice_p2[get_index(jmax,i)];
    }
}

__global__ void poisson_kernel_2(double* cudaDevice_r, double* cudaDevice_p, double* cudaDevice_p2, double* cudaDevice_rhs2, int imax, int jmax, double delx, double dely, double omg){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int j = idx/(jmax+2);
    int i = idx%(jmax+2);
    int eiw,eie,ejs,ejn;
    if(j>=1&&j<jmax+1){
        if(i>=1&&i<imax+1){
            eiw=1;eie=1;ejs=1;ejn=1;
            cudaDevice_p[get_index(j,i)] = (1-omg)*cudaDevice_p2[get_index(j,i)]
            +
            omg/((eie+eiw)/(delx*delx)+(ejn+ejs)/(dely*dely)) * (
                (eie*cudaDevice_p2[get_index(j,i+1)]+eiw*cudaDevice_p2[get_index(j,i-1)])/(delx*delx)
                +(ejn*cudaDevice_p2[get_index(j+1,i)]+ejs*cudaDevice_p2[get_index(j-1,i)])/(dely*dely)
                -cudaDevice_rhs2[get_index(j,i)]
            );

            cudaDevice_r[get_index(j,i)] = (
                eie*(cudaDevice_p2[get_index(j,i+1)]-cudaDevice_p2[get_index(j,i)])
                -eiw*(cudaDevice_p2[get_index(j,i)]-cudaDevice_p2[get_index(j,i-1)])
                )/(delx*delx)
            +    (
                ejn*(cudaDevice_p2[get_index(j+1,i)]-cudaDevice_p2[get_index(j,i)])
                -ejs*(cudaDevice_p2[get_index(j,i)]-cudaDevice_p2[get_index(j-1,i)])
                )/(dely*dely)
            - cudaDevice_rhs2[get_index(j,i)];

            cudaDevice_r[get_index(j,i)] = cudaDevice_r[get_index(j,i)]*cudaDevice_r[get_index(j,i)];
        }
    }
}

int poisson(int imax, int jmax,double delx,double dely,double eps,int itermax,double omg){
    int it;
    double sum;
    double res;
    double* cudaDevice_r;
    hipMalloc(&cudaDevice_r, (imax+2)*(jmax+2) *sizeof(double));
    for(it=0;it<itermax;it++){
        int nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
        /* Init of r to 0 can be moved out of the loop */
        fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_r, (imax+2)*(jmax+2), 0);

        nBlocks = (max(imax,jmax)+2 + THREADSPB-1)/THREADSPB;
        poisson_kernel_1<<<nBlocks, THREADSPB>>>(cudaDevice_p, cudaDevice_p2, cudaDevice_r, imax, jmax);

        nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
        poisson_kernel_2<<<nBlocks, THREADSPB>>>(cudaDevice_r, cudaDevice_p, cudaDevice_p2, cudaDevice_rhs2, imax, jmax, delx, dely, omg);

        sum = sum_vector(cudaDevice_r, (imax+2)*(jmax+2));
        res=sqrt(sum/(imax*jmax));
        if(res<eps){
            break;
        }
        /* copy p to stale p(p2) */
        double* tmp_p = cudaDevice_p2;
        cudaDevice_p2 = cudaDevice_p;
        cudaDevice_p = tmp_p;
    }
    hipFree(cudaDevice_r);
    return it;
}

__global__ void adap_uv_kernel(double* cudaDevice_u, double* cudaDevice_v, double* cudaDevice_f2, double* cudaDevice_g2, double* cudaDevice_p2, int imax,int jmax,double delt,double delx,double dely){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int j = idx/(jmax+2);
    int i = idx%(jmax+2);
    if(j>=1&&j<jmax+1){
        if(i>=1&&i<imax){
            cudaDevice_u[get_index(j,i)] = cudaDevice_f2[get_index(j,i)]-delt/delx*(cudaDevice_p2[get_index(j,i+1)]-cudaDevice_p2[get_index(j,i)]);
        }
    }
    if(j>=1&&j<jmax){
        if(i>=1&&i<imax+1){
            cudaDevice_v[get_index(j,i)] = cudaDevice_g2[get_index(j,i)]-delt/dely*(cudaDevice_p2[get_index(j+1,i)]-cudaDevice_p2[get_index(j,i)]);
        }
    }
}

void adap_uv(int imax, int jmax, double delt, double delx, double dely){
    int nBlocks = nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
    adap_uv_kernel<<<nBlocks, THREADSPB>>>(cudaDevice_u, cudaDevice_v, cudaDevice_f2, cudaDevice_g2, cudaDevice_p2, imax, jmax, delt, delx, dely);
}
