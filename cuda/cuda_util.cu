#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "cuda_util.h"

#define THREADSPB 256

void cuda_init(int imax, int jmax){
}

__global__ void setbound_kernel_x(double** cudaDevice_u, double** cudaDevice_v, double** cudaDevice_u2, double** cudaDevice_v2, int imax, int jmax){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx>=1&&idx<jmax+1){
        cudaDevice_u2[idx][0] = 0;
        cudaDevice_u2[idx][imax] = 0;
        cudaDevice_v2[idx][0] = -cudaDevice_v[idx][1];
        cudaDevice_v2[idx][imax+1] = -cudaDevice_v[idx][imax];
    }
}

__global__ void setbound_kernel_y(double** cudaDevice_u, double** cudaDevice_v, double** cudaDevice_u2, double** cudaDevice_v2, int imax, int jmax){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int us = 1;
    if(idx>=1&&idx<imax+1){
        cudaDevice_v2[0][idx] = 0;
        cudaDevice_v2[jmax][idx] = 0;
        cudaDevice_u2[0][idx] = -cudaDevice_u[1][idx];
        cudaDevice_u2[jmax+1][idx] = 2*us - cudaDevice_u[jmax][idx];
    }
}

void setbound(double **u,double **v,int imax,int jmax,int wW, int wE,int wN,int wS){
    double *cudaDevice_u, *cudaDevice_v, *cudaDevice_p, *cudaDevice_f, *cudaDevice_g, *cudaDevice_rhs;
    double *cudaDevice_u2, *cudaDevice_v2, *cudaDevice_p2, *cudaDevice_f2, *cudaDevice_g2, *cudaDevice_rhs2;
    hipMalloc(&cudaDevice_u, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_v, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_p, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_f, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_g, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_rhs, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_u2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_v2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_p2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_f2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_g2, (imax+2)*(jmax+2)*sizeof(double));
    hipMalloc(&cudaDevice_rhs2, (imax+2)*(jmax+2)*sizeof(double));

    hipMemcpy(cudaDevice_u, u, sizeof(double)*(imax+2)*(jmax+2), hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_v, v, sizeof(double)*(imax+2)*(jmax+2), hipMemcpyHostToDevice);
    int nBlocks = (jmax+1 + THREADSPB-1)/THREADSPB;
    setbound_kernel_x<<<nBlocks, THREADSPB>>>(cudaDevice_u, cudaDevice_v, cudaDevice_u2, cudaDevice_v2,imax,jmax);
    nBlocks = (imax+1 + THREADSPB-1)/THREADSPB;
    setbound_kernel_y<<<nBlocks, THREADSPB>>>(cudaDevice_u, cudaDevice_v, cudaDevice_u2, cudaDevice_v2,imax,jmax);
    hipMemcpy(u, cudaDevice_u2, sizeof(double)*(imax+2)*(jmax+2), hipMemcpyDeviceToHost);
    hipMemcpy(v, cudaDevice_v2, sizeof(double)*(imax+2)*(jmax+2), hipMemcpyDeviceToHost);
    return;
}