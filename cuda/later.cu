#include "hip/hip_runtime.h"
thrust::device_ptr<double> d_ptr = thrust::device_pointer_cast(device_p);
double sum = thrust::reduce(thrust::device, d_ptr, d_ptr+length, (double)0.0, thrust::plus<double>());

__global__ void fg_kernel(double** u, double** v, double** f, double** g, int imax, int jmax, double delt, double delx, double dely, double gx, double gy, double gamma, double Re){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i<imax&&i>=1){
        if(j>=1&&j<jmax+1){
            a=u[j][i]+u[j][i+1];
            b=u[j][i-1]+u[j][i];
            c=u[j][i]-u[j][i+1];
            d=u[j][i-1]-u[j][i];
            e=u[j][i]+u[j+1][i];
            ff=u[j-1][i]+u[j][i];
            gg=u[j][i]-u[j+1][i];
            h=u[j-1][i]-u[j][i];
            va=v[j][i]+v[j][i+1];
            vb=v[j-1][i]+v[j-1][i+1];
            u2x=1/delx*((a/2)*(a/2)-(b/2)*(b/2))
            +gamma*1/delx*(abs(a)/2*c/2-abs(b)/2*d/2);
            uvy=1/dely*(va/2*e/2-vb/2*ff/2)
            +gamma*1/dely*(abs(va)/2*gg/2-abs(vb)/2*h/2);
            u2x2=(u[j][i+1]-2*u[j][i]+u[j][i-1])/(delx*delx);
            u2y2=(u[j+1][i]-2*u[j][i]+u[j-1][i])/(dely*dely);
            f[j][i]=u[j][i]+delt*(1/Re*(u2x2+u2y2)-u2x-uvy+gx);
        }
    }
    if(j>=1&&j<jmax){
        if(i>=i&&i<imax+1){
            a=v[j][i]+v[j][i+1];
            b=v[j][i-1]+v[j][i];
            c=v[j][i]-v[j][i+1];
            d=v[j][i-1]-v[j][i];
            e=v[j][i]+v[j+1][i];
            ff=v[j-1][i]+v[j][i];
            gg=v[j][i]-v[j+1][i];
            h=v[j-1][i]-v[j][i];
            ua=u[j][i]+u[j+1][i];
            ub=u[j][i-1]+u[j+1][i-1];
            uvx=1/delx*(ua/2*a/2-ub/2*b/2)
            +gamma*1/delx*(abs(ua)/2*c/2-abs(ub)/2*d/2);
            v2y=1/dely*((e/2)*(e/2)-(ff/2)*(ff/2))
            +gamma*1/dely*(abs(e)/2*gg/2-abs(ff)/2*h/2);
            v2x2=(v[j][i+1]-2*v[j][i]+v[j][i-1])/(delx*delx);
            v2y2=(v[j+1][i]-2*v[j][i]+v[j-1][i])/(dely*dely);
            g[j][i]=v[j][i]+delt*(1/Re*(v2x2+v2y2)-uvx-v2y+gy);
        }
    }
}

void comp_fg(double **u,double **v,double **f,double **g, int imax,int jmax,double delt,double delx,double dely,double gx,double gy,double gamma,double Re){
    dim3 dimBlock, dimGrid;
    dimBlock.x = BLOCK_SIZE;
    dimBlock.y = BLOCK_SIZE;
    int total = (imax+1)*(jmax+1);
    dimGrid.x = (total + BLOCK_SIZE-1)/BLOCK_SIZE;
    dimGrid.y = (total + BLOCK_SIZE-1)/BLOCK_SIZE;
    int nBlocks = (total + THREADSPB-1)/THREADSPB;

	int j,i;
	double a,b,c,d,e,ff,gg,h,va,vb,u2x,uvy,u2x2,u2y2;
	double ua,ub,uvx,v2y,v2x2,v2y2;
    for(j=0;j<jmax+2;j++){
        for(i=0;i<imax+2;i++){
            f[j][i]=0;
            g[j][i]=0;
        }
    }
	for(j=1;j<jmax+1;j++){
		f[j][0]=u[j][0];
		f[j][imax]=u[j][imax];
	}
	for(i=1;i<imax+1;i++){
		g[0][i]=v[0][i];
		g[jmax][i]=v[jmax][i];
	}
	fg_kernel<<<dimGrid, dimBlock>>>(u,v,f,g,imax,jmax,delt,delx,dely,gx,gy,gamma,Re);
	return;
}

__global__ void rhs_kernel(double** f, double** g, double** rhs, int imax, int jmax, double delt, double delx, double dely){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
	rhs[j][i]=1/delt*((f[j][i]-f[j][i-1])/delx+(g[j][i]-g[j-1][i])/dely);
}

void comp_rhs(double **f, double **g,double **rhs,int imax,int jmax,double delt,double delx,double dely){
	dim3 dimBlock, dimGrid;
    dimBlock.x = BLOCK_SIZE;
    dimBlock.y = BLOCK_SIZE;
    int total = (imax+1)*(jmax+1);
    dimGrid.x = (total + BLOCK_SIZE-1)/BLOCK_SIZE;
    dimGrid.y = (total + BLOCK_SIZE-1)/BLOCK_SIZE;
	rhs_kernel<<<dimGrid, dimBlock>>>(f,g,rhs,imax,jmax,delt,delx,dely);
	return;
}

__global__ void poisson_kernel(double** p, double** rhs, double delx, double dely, double omg){
	int j = 
	int i = 
	eiw=1;eie=1;ejs=1;ejn=1;
	p[j][i]=(1-omg)*p[j][i]
	+omg/((eie+eiw)/(delx*delx)+(ejn+ejs)/(dely*dely))
	*((eie*p[j][i+1]+eiw*p[j][i-1])/(delx*delx)
	+(ejn*p[j+1][i]+ejs*p[j-1][i])/(dely*dely)-rhs[j][i]);

	int tmp =(eie*(p[j][i+1]-p[j][i])-eiw*(p[j][i]-p[j][i-1]))/(delx*delx)
	+(ejn*(p[j+1][i]-p[j][i])-ejs*(p[j][i]-p[j-1][i]))/(dely*dely)-rhs[j][i];
	sum[j][i] = tmp*tmp;
}

int poisson(double **p,double **rhs,int imax,int jmax,double delx,double dely,double eps,int itermax,double omg){
	int it,j,i,eiw,eie,ejs,ejn;
	double **r;
    double res;
    double** sum = RMATRIX(0, jmax+1,0,imax+1);
	for(it=0;it<itermax;it++){
		for(j=1;j<jmax+1;j++){
			p[j][0]=p[j][1];
			p[j][imax+1]=p[j][imax];
		}
		for(i=1;i<imax+1;i++){
			p[0][i]=p[1][i];
			p[jmax+1][i]=p[jmax][i];
		}
		poisson_kernel<<<>>>(p, rhs, imax, jmax, delx, dely, omg);
		int reduce_sum = kernelSum(sum);
        res=sqrt(reduce_sum/(imax*jmax));
        if(res<eps){
            printf("Converged...%f\n",res);
        	break;
        }
	}
	return it;
}

__global__ void adap_kernel(double** u, double** v, double** f, double** g, double** p, double delx, double dely){
	int j = 
	int i = 
	u[j][i]=f[j][i]-delt/delx*(p[j][i+1]-p[j][i]);
	v[j][i]=g[j][i]-delt/dely*(p[j+1][i]-p[j][i]);
}

void adap_uv(double **u,double **v,double **f,double **g,double **p,int imax,int jmax,double delt,double delx,double dely){
    adap_kernel<<<>>>(u,v,f,g,p,imax,jmax,delx,dely);
	return;
}
