#include "hip/hip_runtime.h"
double* tmp_u = cudaDevice_u2;
double* tmp_v = cudaDevice_v2;
double* tmp_p = cudaDevice_p2;
double* tmp_f = cudaDevice_f2;
double* tmp_g = cudaDevice_g2;
double* tmp_rhs = cudaDevice_rhs2;
cudaDevice_u2 = cudaDevice_u;
cudaDevice_v2 = cudaDevice_v;
cudaDevice_p2 = cudaDevice_p;
cudaDevice_f2 = cudaDevice_f;
cudaDevice_g2 = cudaDevice_g;
cudaDevice_rhs2 = cudaDevice_rhs;
cudaDevice_u = tmp_u;
cudaDevice_v = tmp_v;
cudaDevice_p = tmp_p;
cudaDevice_f = tmp_f;
cudaDevice_g = tmp_g;
cudaDevice_rhs = tmp_rhs;
int nBlocks = ((imax+2)*(jmax+2) + THREADSPB-1)/THREADSPB;
fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_u, (imax+2)*(jmax+2), 0);
fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_v, (imax+2)*(jmax+2), 0);
fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_p, (imax+2)*(jmax+2), 0);
fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_f, (imax+2)*(jmax+2), 0);
fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_g, (imax+2)*(jmax+2), 0);
fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_rhs, (imax+2)*(jmax+2), 0);
fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_u2, (imax+2)*(jmax+2), 0);
fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_v2, (imax+2)*(jmax+2), 0);
fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_p2, (imax+2)*(jmax+2), 0);
fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_f2, (imax+2)*(jmax+2), 0);
fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_g2, (imax+2)*(jmax+2), 0);
fill_val<<<nBlocks, THREADSPB>>>(cudaDevice_rhs2, (imax+2)*(jmax+2), 0);

hipblasHandle_t handle;
    hipblasCreate(&handle);
    double mymax = 0.0;
    int max_idx = 0;
    hipblasIdamax(handle, length, device_p, 1, &max_idx);
    double* tmp = (double*)malloc(sizeof(double));
    hipMemcpy(tmp,&(device_p[max_idx]),sizeof(double),hipMemcpyDeviceToHost);
    mymax = *tmp;
    free(tmp);
    return mymax;

void CFDCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {

    int totalBytes = sizeof(float) * 3 * N;
    size_t size = N * sizeof(float);

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_x;
    float* device_y;
    float* device_result;

    // allocate device memory buffers on the GPU using hipMalloc
    hipMalloc(&device_x, size);
    hipMalloc(&device_y, size);
    hipMalloc(&device_result, size);

    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    // copy input arrays to the GPU using hipMemcpy
    hipMemcpy(device_x, xarray, size, hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, size, hipMemcpyHostToDevice);

    // run kernel
    double kernelStartTime = CycleTimer::currentSeconds();
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);
    hipDeviceSynchronize();
    double kernelEndTime = CycleTimer::currentSeconds();

    // copy result from GPU using hipMemcpy
    hipMemcpy(resultarray, device_result, size, hipMemcpyDeviceToHost);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));
    printf("Kernel run time: %.3f ms\n", 1000.f * (kernelEndTime-kernelStartTime));
    
    // free memory buffers on the GPU
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
}

void printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);
    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}

template<typename T>
__global__ 
void kernelSum(const T* __restrict__ input, 
        const size_t lda, // pitch of input in words of sizeof(T)
        T* __restrict__ per_block_results, 
                const size_t n)
{
    extern __shared__ T sdata[];

    T x = 0.0;
    const T * p = &input[blockIdx.x * lda];
    // Accumulate per thread partial sum
    for(int i=threadIdx.x; i < n; i += blockDim.x) {
        x += p[i];
    }

    // load thread partial sum into shared memory
    sdata[threadIdx.x] = x;
    __syncthreads();

    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if(threadIdx.x < offset) {
            sdata[threadIdx.x] += sdata[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // thread 0 writes the final result
    if(threadIdx.x == 0) {
        per_block_results[blockIdx.x] = sdata[0];
    }
}

