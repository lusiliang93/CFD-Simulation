#include "hip/hip_runtime.h"
void CFDCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {

    int totalBytes = sizeof(float) * 3 * N;
    size_t size = N * sizeof(float);

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_x;
    float* device_y;
    float* device_result;

    // allocate device memory buffers on the GPU using hipMalloc
    hipMalloc(&device_x, size);
    hipMalloc(&device_y, size);
    hipMalloc(&device_result, size);

    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    // copy input arrays to the GPU using hipMemcpy
    hipMemcpy(device_x, xarray, size, hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, size, hipMemcpyHostToDevice);

    // run kernel
    double kernelStartTime = CycleTimer::currentSeconds();
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);
    hipDeviceSynchronize();
    double kernelEndTime = CycleTimer::currentSeconds();

    // copy result from GPU using hipMemcpy
    hipMemcpy(resultarray, device_result, size, hipMemcpyDeviceToHost);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));
    printf("Kernel run time: %.3f ms\n", 1000.f * (kernelEndTime-kernelStartTime));
    
    // free memory buffers on the GPU
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
}

void printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);
    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}

template<typename T>
__global__ 
void kernelSum(const T* __restrict__ input, 
        const size_t lda, // pitch of input in words of sizeof(T)
        T* __restrict__ per_block_results, 
                const size_t n)
{
    extern __shared__ T sdata[];

    T x = 0.0;
    const T * p = &input[blockIdx.x * lda];
    // Accumulate per thread partial sum
    for(int i=threadIdx.x; i < n; i += blockDim.x) {
        x += p[i];
    }

    // load thread partial sum into shared memory
    sdata[threadIdx.x] = x;
    __syncthreads();

    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if(threadIdx.x < offset) {
            sdata[threadIdx.x] += sdata[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // thread 0 writes the final result
    if(threadIdx.x == 0) {
        per_block_results[blockIdx.x] = sdata[0];
    }
}

