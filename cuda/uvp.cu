#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <math.h>
#include "init.h"
#include "uvp.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

extern float toBW(int bytes, float sec);

void printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);
    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}

template<typename T>
__global__ 
void kernelSum(const T* __restrict__ input, 
        const size_t lda, // pitch of input in words of sizeof(T)
        T* __restrict__ per_block_results, 
                const size_t n)
{
    extern __shared__ T sdata[];

    T x = 0.0;
    const T * p = &input[blockIdx.x * lda];
    // Accumulate per thread partial sum
    for(int i=threadIdx.x; i < n; i += blockDim.x) {
        x += p[i];
    }

    // load thread partial sum into shared memory
    sdata[threadIdx.x] = x;
    __syncthreads();

    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if(threadIdx.x < offset) {
            sdata[threadIdx.x] += sdata[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // thread 0 writes the final result
    if(threadIdx.x == 0) {
        per_block_results[blockIdx.x] = sdata[0];
    }
}

void CDFCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {

    int totalBytes = sizeof(float) * 3 * N;
    size_t size = N * sizeof(float);

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_x;
    float* device_y;
    float* device_result;

    // allocate device memory buffers on the GPU using hipMalloc
    hipMalloc(&device_x, size);
    hipMalloc(&device_y, size);
    hipMalloc(&device_result, size);

    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    // copy input arrays to the GPU using hipMemcpy
    hipMemcpy(device_x, xarray, size, hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, size, hipMemcpyHostToDevice);

    // run kernel
    double kernelStartTime = CycleTimer::currentSeconds();
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);
    hipDeviceSynchronize();
    double kernelEndTime = CycleTimer::currentSeconds();

    // copy result from GPU using hipMemcpy
    hipMemcpy(resultarray, device_result, size, hipMemcpyDeviceToHost);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));
    printf("Kernel run time: %.3f ms\n", 1000.f * (kernelEndTime-kernelStartTime));
    
    // free memory buffers on the GPU
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
}

__global__ void fg_kernel(double** u, double** v, double** f, double** g, int imax, int jmax, double delt, double delx, double dely, double gx, double gy, double gamma, double Re){
	int j = 
	int i = 
	// Note: loop start and stop conditions are different
	a=u[j][i]+u[j][i+1];
	b=u[j][i-1]+u[j][i];
	c=u[j][i]-u[j][i+1];
	d=u[j][i-1]-u[j][i];
	e=u[j][i]+u[j+1][i];
	ff=u[j-1][i]+u[j][i];
	gg=u[j][i]-u[j+1][i];
	h=u[j-1][i]-u[j][i];
	va=v[j][i]+v[j][i+1];
	vb=v[j-1][i]+v[j-1][i+1];
	u2x=1/delx*((a/2)*(a/2)-(b/2)*(b/2))
	+gamma*1/delx*(abs(a)/2*c/2-abs(b)/2*d/2);
	uvy=1/dely*(va/2*e/2-vb/2*ff/2)
	+gamma*1/dely*(abs(va)/2*gg/2-abs(vb)/2*h/2);
	u2x2=(u[j][i+1]-2*u[j][i]+u[j][i-1])/(delx*delx);
	u2y2=(u[j+1][i]-2*u[j][i]+u[j-1][i])/(dely*dely);
	f[j][i]=u[j][i]+delt*(1/Re*(u2x2+u2y2)-u2x-uvy+gx);

	a=v[j][i]+v[j][i+1];
	b=v[j][i-1]+v[j][i];
	c=v[j][i]-v[j][i+1];
	d=v[j][i-1]-v[j][i];
	e=v[j][i]+v[j+1][i];
	ff=v[j-1][i]+v[j][i];
	gg=v[j][i]-v[j+1][i];
	h=v[j-1][i]-v[j][i];
	ua=u[j][i]+u[j+1][i];
	ub=u[j][i-1]+u[j+1][i-1];
	uvx=1/delx*(ua/2*a/2-ub/2*b/2)
	+gamma*1/delx*(abs(ua)/2*c/2-abs(ub)/2*d/2);
	v2y=1/dely*((e/2)*(e/2)-(ff/2)*(ff/2))
	+gamma*1/dely*(abs(e)/2*gg/2-abs(ff)/2*h/2);
	v2x2=(v[j][i+1]-2*v[j][i]+v[j][i-1])/(delx*delx);
	v2y2=(v[j+1][i]-2*v[j][i]+v[j-1][i])/(dely*dely);
	g[j][i]=v[j][i]+delt*(1/Re*(v2x2+v2y2)-uvx-v2y+gy);
}

void comp_fg(double **u,double **v,double **f,double **g, int imax,int jmax,double delt,double delx,double dely,double gx,double gy,double gamma,double Re){
	int j,i;
	double a,b,c,d,e,ff,gg,h,va,vb,u2x,uvy,u2x2,u2y2;
	double ua,ub,uvx,v2y,v2x2,v2y2;
    for(j=0;j<jmax+2;j++){
        for(i=0;i<imax+2;i++){
            f[j][i]=0;
            g[j][i]=0;
        }
    }
	for(j=1;j<jmax+1;j++){
		f[j][0]=u[j][0];
		f[j][imax]=u[j][imax];
	}
	for(i=1;i<imax+1;i++){
		g[0][i]=v[0][i];
		g[jmax][i]=v[jmax][i];
	}
	fg_kernel<<<>>>(u,v,f,g,imax,jmax,delt,delx,dely,gx,gy,gamma,Re);
	return;
}

__global__ void rhs_kernel(double** f, double** g, double** rhs, int imax, int jmax, double delt, double delx, double dely){
	int j = 
	int i = 
	rhs[j][i]=1/delt*((f[j][i]-f[j][i-1])/delx+(g[j][i]-g[j-1][i])/dely);
}

void comp_rhs(double **f, double **g,double **rhs,int imax,int jmax,double delt,double delx,double dely){
	int j,i;
    for(j=0;j<jmax+2;j++){
        for(i=0;i<imax+2;i++){
            rhs[j][i]=0;
        }
    }
	rhs_kernel<<<>>>(f,g,rhs,imax,jmax,delt,delx,dely);
	return;
}

__global__ void poisson_kernel(double** p, double** rhs, double delx, double dely, double omg){
	int j = 
	int i = 
	eiw=1;eie=1;ejs=1;ejn=1;
	p[j][i]=(1-omg)*p[j][i]
	+omg/((eie+eiw)/(delx*delx)+(ejn+ejs)/(dely*dely))
	*((eie*p[j][i+1]+eiw*p[j][i-1])/(delx*delx)
	+(ejn*p[j+1][i]+ejs*p[j-1][i])/(dely*dely)-rhs[j][i]);

	int tmp =(eie*(p[j][i+1]-p[j][i])-eiw*(p[j][i]-p[j][i-1]))/(delx*delx)
	+(ejn*(p[j+1][i]-p[j][i])-ejs*(p[j][i]-p[j-1][i]))/(dely*dely)-rhs[j][i];
	sum[j][i] = tmp*tmp;
}

int poisson(double **p,double **rhs,int imax,int jmax,double delx,double dely,double eps,int itermax,double omg){
	int it,j,i,eiw,eie,ejs,ejn;
	double **r;
    double res;
    double** sum = RMATRIX(0, jmax+1,0,imax+1);
	for(it=0;it<itermax;it++){
		for(j=1;j<jmax+1;j++){
			p[j][0]=p[j][1];
			p[j][imax+1]=p[j][imax];
		}
		for(i=1;i<imax+1;i++){
			p[0][i]=p[1][i];
			p[jmax+1][i]=p[jmax][i];
		}
		poisson_kernel<<<>>>(p, rhs, imax, jmax, delx, dely, omg);
		int reduce_sum = kernelSum(sum);
        res=sqrt(reduce_sum/(imax*jmax));
        if(res<eps){
            printf("Converged...%f\n",res);
        	break;
        }
	}
	return it;
}

__global__ void adap_kernel(double** u, double** v, double** f, double** g, double** p, double delx, double dely){
	int j = 
	int i = 
	u[j][i]=f[j][i]-delt/delx*(p[j][i+1]-p[j][i]);
	v[j][i]=g[j][i]-delt/dely*(p[j+1][i]-p[j][i]);
}

void adap_uv(double **u,double **v,double **f,double **g,double **p,int imax,int jmax,double delt,double delx,double dely){
    adap_kernel<<<>>>(u,v,f,g,p,imax,jmax,delx,dely);
	return;
}
